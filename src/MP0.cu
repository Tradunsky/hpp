
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#if defined(_MSC_VER) || defined(__MINGW32__) //__MINGW32__ should goes before __GNUC__
  #define JL_SIZE_T_SPECIFIER    "%Iu"
  #define JL_SSIZE_T_SPECIFIER   "%Id"
  #define JL_PTRDIFF_T_SPECIFIER "%Id"
#elif defined(__GNUC__)
  #define JL_SIZE_T_SPECIFIER    "%zu"
  #define JL_SSIZE_T_SPECIFIER   "%zd"
  #define JL_PTRDIFF_T_SPECIFIER "%zd"
#else
  // TODO figure out which to use.
  #if NUMBITS == 32
    #define JL_SIZE_T_SPECIFIER    something_unsigned
    #define JL_SSIZE_T_SPECIFIER   something_signed
    #define JL_PTRDIFF_T_SPECIFIER something_signed
  #else
    #define JL_SIZE_T_SPECIFIER    something_bigger_unsigned
    #define JL_SSIZE_T_SPECIFIER   something_bigger_signed
    #define JL_PTRDIFF_T_SPECIFIER something-bigger_signed
  #endif
#endif

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__device__ unsigned int deviceFun(unsigned int number) {
	return number;
}

/**
 * CUDA kernel function that reverses the order of bits in each element of the array.
 */
__global__ void cudaFun(void *data) {
	unsigned int *idata = (unsigned int*) data;
	idata[threadIdx.x] = deviceFun(idata[threadIdx.x]);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main2(void) {
	printf("Program begin...");
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("\nDevice count: %i", deviceCount);
	for (int dev = 0; dev < deviceCount; dev++) {
		hipDeviceProp_t deviceProp;
		CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProp, dev));
		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				printf("\nNo CUDA GPU has been detected");
				return -1;
			} else if (deviceCount == 1) {
				//@@ WbLog is a provided logging API (similar to Log4J).
				//@@ The logging function wbLog takes a level which is either
				//@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
				//@@ message to be printed.
				printf("\nThere is 1 device supporting CUDA");
			} else {
				printf("\nThere are %i devices supporting CUDA", deviceCount);
			}
		}
		printf("\nDevice %i name: %s", dev, deviceProp.name);
		printf("\nComputational Capabilities: %i.%i",deviceProp.major,deviceProp.minor);
		printf("\nMaximum global memory size: "JL_SIZE_T_SPECIFIER, deviceProp.totalGlobalMem);
		printf("\nMaximum constant memory size: "JL_SIZE_T_SPECIFIER, deviceProp.totalConstMem);
		printf("\nMaximum shared memory size per block: %i",deviceProp.sharedMemPerBlock);
		printf("\nMaximum threads per block: %i",deviceProp.maxThreadsPerBlock);
		printf("\nMaximum block dimensions: %ix%ix%i", deviceProp.maxThreadsDim[0],
													deviceProp.maxThreadsDim[1],
													deviceProp.maxThreadsDim[2]);
		printf("\nMaximum grid dimensions: %ix%ix%i", deviceProp.maxGridSize[0],
												   deviceProp.maxGridSize[1],
												   deviceProp.maxGridSize[2]);
		printf("\nWarp size: %i",deviceProp.warpSize);
	}
	printf("\nProgram end...");
	return 0;
}
